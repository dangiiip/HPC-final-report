
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o password_Dangi password_crack_Dangi.cu


     To Run:
     ./password_Dangi > resultpwd_cuda_Dangi.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char Dangi_password1[] = "PD6067";
  char Dangi_password2[] = "CR2056";
  char Dangi_password3[] = "KN1972";
  char Dangi_password4[] = "IC2019";

  char *c = attempt;
  char *r = attempt;
  char *k = attempt;
  char *n = attempt;
  char *k1 = Dangi_password1;
  char *k2 = Dangi_password2;
  char *k3 = Dangi_password3;
  char *k4 = Dangi_password4;

  while(*c == *k1) {
   if(*c == '\0')
    {
    printf("Password: %s\n",Dangi_password1);
      break;
    }

    c++;
    k1++;
  }
    
  while(*r == *k2) {
   if(*r == '\0')
    {
    printf("Password: %s\n",Dangi_password2);
      break;
}

    r++;
    k2++;
  }

  while(*k == *k3) {
   if(*k == '\0')
    {
    printf("Password: %s\n",Dangi_password3);
      break;
    }

    k++;
    k3++;
  }

  while(*n == *k4) {
   if(*n == '\0')
    {
    printf("Password: %s\n",Dangi_password4);
      return 1;
    }

    n++;
    k4++;
  }
  return 0;

}
__global__ void  kernel() {
char N,a,v,s;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(N='0'; N<='9'; N++){
      for(a='0'; a<='9'; a++){
        for(v='0';v<='9';v++){
          for(s='0';s<='9';s++){
            password[2] = N;
            password[3] = a;
            password[4]= v;
            password[5]=s;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



